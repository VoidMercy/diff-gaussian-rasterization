#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Assuming declaration of ray_render_composing is available
__device__ float3 ray_render_composing(int x, int y, const int N_GAUSSIANS, int *gaussians, float *depths, float2 *mean2D, float *cov2D, const float* colors_precomp, float4* conic_opacity);

// Random state setup kernel
__global__ void setup_kernel(hiprandState *state, int seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}

// Test kernel for ray_render_composing
__global__ void testRayRenderComposing(hiprandState *states, int *gaussians, float *depths, float2 *mean2D, float *cov2D, float *colors_precomp, float4 *conic_opacity, int N_GAUSSIANS) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N_GAUSSIANS) {
        // Random values setup
        depths[idx] = hiprand_uniform(&states[idx]) * 100.0f;  // Random depths between 0 and 100
        mean2D[idx] = make_float2(hiprand_uniform(&states[idx]) * 800, hiprand_uniform(&states[idx]) * 600);  // Random positions within 800x600 window
        cov2D[idx] = hiprand_uniform(&states[idx]) * 10.0f + 1.0f;  // Random covariances between 1 and 11
        int colorIndex = idx * 3;
        colors_precomp[colorIndex] = hiprand_uniform(&states[idx]);  // Random color R
        colors_precomp[colorIndex + 1] = hiprand_uniform(&states[idx]);  // Random color G
        colors_precomp[colorIndex + 2] = hiprand_uniform(&states[idx]);  // Random color B
        conic_opacity[idx] = make_float4(1.0f, 0.0f, 1.0f, hiprand_uniform(&states[idx]));  // Random opacity
    }

    // Ensure all data is generated before testing
    __syncthreads();

    // Run the ray rendering composing test on a specific pixel
    if (idx == 0) {
        int testX = 400, testY = 300;
        float3 result = ray_render_composing(testX, testY, N_GAUSSIANS, gaussians, depths, mean2D, cov2D, colors_precomp, conic_opacity);
        printf("Accumulated color at (%d, %d): R=%f, G=%f, B=%f\n", testX, testY, result.x, result.y, result.z);
    }
}

int main() {
    const int N_GAUSSIANS = 1000;
    int *gaussians;
    float *depths;
    float2 *mean2D;
    float *cov2D;
    float *colors_precomp;
    float4 *conic_opacity;
    hiprandState *states;

    // Allocate memory for arrays and states
    hipMalloc(&gaussians, N_GAUSSIANS * sizeof(int));
    hipMalloc(&depths, N_GAUSSIANS * sizeof(float));
    hipMalloc(&mean2D, N_GAUSSIANS * sizeof(float2));
    hipMalloc(&cov2D, N_GAUSSIANS * sizeof(float));
    hipMalloc(&colors_precomp, N_GAUSSIANS * 3 * sizeof(float));
    hipMalloc(&conic_opacity, N_GAUSSIANS * sizeof(float4));
    hipMalloc(&states, N_GAUSSIANS * sizeof(hiprandState));

    // Initialize indices
    for (int i = 0; i < N_GAUSSIANS; i++) {
        gaussians[i] = i;
    }

    // Setup random states
    setup_kernel<<<(N_GAUSSIANS + 255) / 256, 256>>>(states, time(NULL));
    hipDeviceSynchronize();

    // Run test
    testRayRenderComposing<<<(N_GAUSSIANS + 255) / 256, 256>>>(states, gaussians, depths, mean2D, cov2D, colors_precomp, conic_opacity, N_GAUSSIANS);
    hipDeviceSynchronize();

    // Clean up
    hipFree(gaussians);
    hipFree(depths);
    hipFree(mean2D);
    hipFree(cov2D);
    hipFree(colors_precomp);
    hipFree(conic_opacity);
    hipFree(states);

    return 0;
}
